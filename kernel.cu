#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void preScanKernel(float *inout, unsigned size, float *sum)
{
    // Perform a local scan on 2*BLOCK_SIZE items
    
    __shared__ float temp[BLOCK_SIZE * 2];
    
    int location = 0;
    int thread = threadIdx.x;
    int index = 2 * blockIdx.x * blockDim.x;
    
    if(index + thread < size)
        temp[thread] = inout[index + thread];
    else
        temp[thread] = 0;
        
    if(index + thread + blockDim.x < size)
        temp[thread + blockDim.x] = inout[index + thread + blockDim.x];
    else
        temp[thread + blockDim.x] = 0;
        
    __syncthreads();
    
    int offset = 1;
    while(offset <= blockDim.x) {
        location = (thread + 1) * 2 * offset - 1;
        if(location < (2 * BLOCK_SIZE))
            temp[location] += temp[location - offset];
        
        offset *= 2;
        __syncthreads();
    }
    
    if(thread == 0) {
        if(sum != NULL)
            sum[blockIdx.x] = temp[2 * blockDim.x - 1];
        
        temp[2 * blockDim.x - 1] = 0;
    }
    
    __syncthreads();
    
    location = 0;
    float val = 0;
    offset = blockDim.x;
    
    while(offset > 0) {
        location = (2 * offset * (thread + 1)) - 1;
        if(location < 2 * BLOCK_SIZE) {
            val = temp[location];
            temp[location] += temp[location - offset];
            temp[location - offset] = val;
        }
        offset >>= 1;
        __syncthreads();
    }
    
    if(index + thread < size)
        inout[index + thread] = temp[thread];
    
    if(index + thread + blockDim.x < size)
        inout[index + thread + blockDim.x] = temp[thread + blockDim.x];
}

__global__ void addKernel(float *inout, float *sum, unsigned size)
{
    // Use the scan of partial sums to update 2*BLOCK_SIZE items
    
    int block = blockIdx.x;
    int thread = threadIdx.x;
    int location = 2 * blockDim.x * block + thread;
    
    if(location < 2 * BLOCK_SIZE)
        inout[location] += sum[blockIdx.x];
        
    if(location + blockDim.x < 2 * BLOCK_SIZE)
        inout[location + blockDim.x] += sum[blockIdx.x];
}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void preScan(float *inout, unsigned in_size)
{
	float *sum;
	unsigned num_blocks;
	hipError_t cuda_ret;
	dim3 dim_grid, dim_block;

	num_blocks = in_size/(BLOCK_SIZE*2);
	if(in_size%(BLOCK_SIZE*2) !=0) num_blocks++;

	dim_block.x = BLOCK_SIZE; dim_block.y = 1; dim_block.z = 1;
	dim_grid.x = num_blocks; dim_grid.y = 1; dim_grid.z = 1;

	if(num_blocks > 1) {
		cuda_ret = hipMalloc((void**)&sum, num_blocks*sizeof(float));
		if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

		preScanKernel<<<dim_grid, dim_block>>>(inout, in_size, sum);
		preScan(sum, num_blocks);
		addKernel<<<dim_grid, dim_block>>>(inout, sum, in_size);

		hipFree(sum);
	}
	else
		preScanKernel<<<dim_grid, dim_block>>>(inout, in_size, NULL);
}